#include "header.h"
#include "cpu.cu"
#include "gpu.cu"

// ================== Function: main ====================
// driver function for the program
int main() {

    // CPU MEMORY ALLOCATION
    int *numOfVertices = (int *) malloc(sizeof(int)); // # of vertices in the graph
    int *arrayLength = (int *) malloc(sizeof(int));

    // PROMPT USER FOR # OF VERTICES
    printf("Enter the number of vertices for graph: ");
    scanf("%d", numOfVertices);

    *arrayLength = *numOfVertices * *numOfVertices;

    // CPU MEMORY
    float* graph = (float *) malloc(*arrayLength * sizeof(float));
    float* result = (float *) malloc(*numOfVertices * sizeof(float));
        
    // FUNCTION CALLS (CPU)
    createGraph(graph, *arrayLength); // Generate the graph & store in array
    printGraph(graph, *arrayLength); // Print the array
    dijkstra(graph, 1, *numOfVertices, result); 
    printSolution(result, *numOfVertices);
    

    // GPU MEMORY ALLOCATION
    int *d_numOfVertices, *d_arrayLength;
    float* d_graph, d_result;

    hipMalloc((void **) &d_numOfVertices, sizeof(int));
    hipMalloc((void **) &d_arrayLength, sizeof(int));
    hipMalloc((void **) &d_graph, (*arrayLength * sizeof(float)));
    hipMalloc((void **) &d_result, (*numOfVertices * sizeof(float)));

    // COPY CPU MEM --> GPU MEM
    hipMemcpy(d_numOfVertices, &numOfVertices, (*arrayLength * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_arrayLength, &arrayLength, (*arrayLength * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_graph, &graph, (*arrayLength * sizeof(float)), hipMemcpyHostToDevice);
    
    

    // FREE GPU MEM
    hipFree(d_graph);
    hipFree(d_numOfVertices);
    hipFree(d_arrayLength);


    // FREE CPU MEM
    free(numOfVertices);
    free(arrayLength);
    free(graph);
    free(result);
    
    return 0;
}
