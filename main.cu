#include "header.h"
#include "cpu.cu"
#include "gpu.cu"

// ================== Function: main ====================
// driver function for the program
int main() {

    // FETCH INPUT FROM USER
    int numOfVertices; // # of vertices in the graph
    printf("Please enter the number of vertices for graph: ");
    scanf("%d", &numOfVertices);
    const int arrayLength = numOfVertices * numOfVertices;

    float* result = (float *) malloc(sizeof(float) * numOfVertices);
    // HOST COPY
    float* graph = (float *) malloc(arrayLength * sizeof(float));

    // DEVICE COPY
    // float* d_graph;
    // allocate memory for the graph
    // hipMalloc((void **) &d_graph, (arrayLength * sizeof(float)));

    // FUNCTION CALLS (CPU)
    createGraph(graph, arrayLength); // Generate the graph & store in array
    printGraph(graph, arrayLength); // Print the array
    dijkstra(graph, 1, numOfVertices, result); 
    printSolution(result, numOfVertices);
    
    // FUNCTION CALLS (GPU)
    /* hipMemcpy(d_graph, &graph, (arrayLength * sizeof(float)), hipMemcpyHostToDevice); */
    /* hipFree(d_graph); */
    return 0;
}
