#include "header.h"
#include "cpu.cu"
#include "gpu.cu"

// ================== Function: main ====================
// driver function for the program
int main() {

    // FETCH INPUT FROM USER
    int *numOfVertices = (int *) malloc(sizeof(int)); // # of vertices in the graph
    int *arrayLength = (int *) malloc(sizeof(int));

    printf("Please enter the number of vertices for graph: ");
    scanf("%d", numOfVertices);

    *arrayLength = *numOfVertices * *numOfVertices;

    float* graph = (float *) malloc(*arrayLength * sizeof(float));
    float* result = (float *) malloc(sizeof(float) * *numOfVertices);
    
    

    // FUNCTION CALLS (CPU)
    createGraph(graph, *arrayLength); // Generate the graph & store in array
    printGraph(graph, *arrayLength); // Print the array
    dijkstra(graph, 1, *numOfVertices, result); 
    printSolution(result, *numOfVertices);
    

    // FUNCTION CALLS (GPU)
    int *d_numOfVertices, *d_arrayLength;
    float* d_graph; // GRAPH DEVICE COPY

    hipMalloc((void **) &d_numOfVertices, sizeof(int));
    hipMalloc((void **) &d_arrayLength, sizeof(int));
    hipMalloc((void **) &d_graph, (*arrayLength * sizeof(float))); // allocate mem

    hipMemcpy(d_numOfVertices, &numOfVertices, (*arrayLength * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_arrayLength, &arrayLength, (*arrayLength * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_graph, &graph, (*arrayLength * sizeof(float)), hipMemcpyHostToDevice);
    
    
    // GPU Dijkstra Call

    hipFree(d_graph);
    hipFree(d_numOfVertices);
    hipFree(d_arrayLength);


    free(numOfVertices);
    free(arrayLength);
    free(graph);
    free(result);
    
    return 0;
}
