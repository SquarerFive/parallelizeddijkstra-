#include "header.h"
#include "cpu.cu"
#include "gpu.cu"


int main() {

    /**************************** TAKE USER INPUT *****************************/
    
    int *numOfVertices = (int *) malloc(sizeof(int));
    int *arrayLength = (int *) malloc(sizeof(int));

    // PROMPT USER FOR # OF VERTICES
    printf("Enter the number of vertices for graph: ");
    scanf("%d", numOfVertices);

    // WILL BE AN ARRAY REPRESENTATION OF A MATRIX
    *arrayLength = *numOfVertices * *numOfVertices;


    /***************************** CPU DIJKSTRA  ******************************/

    // ALLOCATE CPU MEMORY
    float* graph = (float *) malloc(*arrayLength * sizeof(float));
    float* result = (float *) malloc(*arrayLength * sizeof(float));

    createGraph(graph, *arrayLength); // Generate the graph & store in array
    printGraph(graph, *arrayLength); // Print the array

    for(int j = 0; j<*numOfVertices; j++) {
        dijkstra(graph, j, *numOfVertices, result); 
        printSolution(result, *numOfVertices);
    }

    /***************************** GPU DIJKSTRA  ******************************/
    
    // initialize the varibles needed in the gpu
    float *d_graph, *d_result;
    bool *d_visited;

    // allocate memory in the gpu for our variables
    hipMalloc((void **) &d_graph, (*arrayLength * sizeof(float)));
    hipMalloc((void **) &d_result, (*arrayLength * sizeof(float)));
    hipMalloc((void **) &d_visited, (*arrayLength * sizeof(bool)));
    
    // copy graph generated in the cpu to the gpu
    hipMemcpy(d_graph, graph, (*arrayLength * sizeof(float)), hipMemcpyHostToDevice);

    // set up the graph using multiple cores & threads
    gpu_setUpGraph<<<*numOfVertices,*numOfVertices>>>(d_result, d_visited);

    // perform dijstra on ALL vertices as src vertex using multiple cores
    gpu_dijkstra<<<*numOfVertices,1>>>(d_graph,d_result, d_visited, *numOfVertices);
   
    // copy the results back to cpu
    hipMemcpy(result, d_result, (*arrayLength * sizeof(float)), hipMemcpyDeviceToHost);
    printGraph(result, *arrayLength);

    
    // free the gpu memory
    hipFree(d_graph);
    hipFree(d_result);
    hipFree(d_visited);

 
    // free the cpu memory
    free(numOfVertices);
    free(arrayLength);
    free(graph);
    free(result);
  
    return 0;
}
